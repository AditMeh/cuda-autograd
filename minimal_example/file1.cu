#include "hip/hip_runtime.h"
#include <stdio.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

__global__ void mykernel(){
  printf("Hello from mykernel\n");
}

void hello(){
  mykernel<<<1,1>>>();
  hipDeviceSynchronize();
}


PYBIND11_MODULE(example, m) {
    m.doc() = "pybind11 example plugin"; // optional module docstring
    m.def("hello", &hello, "A function that adds two numbers");
}