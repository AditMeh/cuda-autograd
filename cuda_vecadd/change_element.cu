#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <iostream>
namespace py = pybind11;

// __global__ void matmulkernel(float*vec1, float* vec2, float *result){
//     int i = threadIdx.x;
//     result[i] = vec1[i] + vec2[i]; // vec addition
// }

void change_element(py::array_t<double> input1) {
    py::buffer_info buf1 = input1.request();

    if (buf1.ndim != 1)
        throw std::runtime_error("Number of dimensions must be one");

    // std::cout << buf1.size << std::endl;
    double *ptr1 =  reinterpret_cast<double*>(buf1.ptr);

    ptr1[0] = 30;
}

PYBIND11_MODULE(change_element, m) {
    m.def("change_element", &change_element, "Add two NumPy arrays");
}