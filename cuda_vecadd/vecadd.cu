#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

namespace py = pybind11;

__global__ void vecaddkernel(double*vec1, double* vec2, double *result){
    int i = threadIdx.x;
    result[i] = vec1[i] + vec2[i]; // vec addition
}

py::array_t<double> vec_add(py::array_t<double> input1, py::array_t<double> input2) {
    py::buffer_info buf1 = input1.request();
    py::buffer_info buf2 = input2.request();

    // Output buffer
    auto result = py::array_t<double>(buf1.size);
    py::buffer_info buf3 = result.request();

    if (buf1.ndim != 1 || buf2.ndim != 1)
        throw std::runtime_error("Number of dimensions must be one");

    if (buf1.size != buf2.size)
        throw std::runtime_error("Input shapes must match");


    double* vec1d; double* vec2d; double* resultd; // the d means device

    int len = buf1.size;

    hipMalloc((void **) &vec1d, len * sizeof(double));
    hipMemcpy((void*)vec1d, buf1.ptr, len * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void **) &vec2d, len * sizeof(double));
    hipMemcpy(vec2d, buf2.ptr, len * sizeof(double), hipMemcpyHostToDevice);
    
    hipMalloc((void **) &resultd, len * sizeof(double));

    dim3 dimBlock(len);
    dim3 dimGrid(1);
    
    vecaddkernel<<<dimGrid, dimBlock>>>(vec1d, vec2d, resultd);
    hipMemcpy(buf3.ptr, resultd, len * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(vec1d); hipFree(vec2d); hipFree(resultd);
    return result;
}

PYBIND11_MODULE(vecadd, m) {
    m.def("vec_add", &vec_add, "Add two NumPy arrays");
}